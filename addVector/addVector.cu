
#include <stdio.h>

#include <hip/hip_runtime.h>


__global__ void add(const int *a, const int *b, int *c, int size){
 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < size){
     c[i] = a[i] + b[i];  
    }
}

int main(void)
{
        int devID=0;
    hipDeviceProp_t props;

    // This will pick the best possible CUDA capable device
    // devID = findCudaDevice(argc, (const char **)argv);

    //Get GPU information
    hipGetDevice(&devID);
    hipGetDeviceProperties(&props, devID);
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",
           devID, props.name, props.major, props.minor);

    printf("printf() is called. Output:\n\n");
    
    int num = 50000;
    unsigned long size = num * sizeof(int);
    int A[num], B[num], C[num];
    
    
    for (int i=0; i < num; ++i)
    {
      A[i] =  i;
      B[i] =  i;
    }
    
    int *gpuA, *gpuB, *gpuC;
    
    hipMalloc((void **)&gpuA, size);
    hipMalloc((void **)&gpuB, size);
    hipMalloc((void **)&gpuC, size);
    
    
    hipMemcpy(gpuA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(gpuB, B, size, hipMemcpyHostToDevice);
    
    
    int threadsPerBlock = 256; // 每个block拥有的线层数量
    int blocksPerGird = (num + threadsPerBlock -1)/threadsPerBlock;  // 多少个block
    
    add<<<blocksPerGird, threadsPerBlock>>>(gpuA, gpuB, gpuC, num);
    
    
    hipMemcpy(C, gpuC, size, hipMemcpyDeviceToHost);
    
    hipFree(gpuA);
    hipFree(gpuB);
    hipFree(gpuC);


    for(int i=0; i < 10; ++i){
      printf("%d ", C[i]); 
    }
    
    return 0;
}
